#include "hip/hip_runtime.h"
//********************************************************************************************
//NOTE: to make this work, also need to add D:\CUDA\lib\x64 to LIB env. variable in BUILD config
//********************************************************************************************

#include <chrono>
#include <iostream>
#include "vector3d.h"
#include "simpoints.h"

// helper functions for cleaner time measuring code
std::chrono::time_point<std::chrono::high_resolution_clock> now() {
    return std::chrono::high_resolution_clock::now();
}
template <typename T>
double milliseconds(T t) {
    return (double) std::chrono::duration_cast<std::chrono::nanoseconds>(t).count() / 1000000;
}


//__constant__ Transducer d_transducers[SimPoints::NumTransducers];
__constant__ uint8_t d_transducers[SimPoints::NumTransducers * sizeof(Transducer)];

__global__
void simKernel(hipPitchedPtr valuesPitch) {
    Transducer *transducers = (Transducer*)d_transducers;

    size_t pitchX = valuesPitch.pitch/sizeof(float);
    float* values = (float*)valuesPitch.ptr;

    int yIdx= threadIdx.x;
    int xIdx= blockIdx.x;
    int zIdx= blockIdx.y;

    int idx = yIdx*SimPoints::NumSimPointsZ*pitchX + zIdx*pitchX + xIdx;

    Vector3D simPt(
                xIdx*SimPoints::SimInterval-(SimPoints::SimExtentsX-1)/2,
                yIdx*SimPoints::SimInterval,
                zIdx*SimPoints::SimInterval-(SimPoints::SimExtentsZ-1)/2
            );

    float reAcc=0;
    float imAcc=0;

    for(int t=0; t< SimPoints::NumTransducers; t++){
        Vector3D delta = simPt - transducers[t].pt;
        float r = delta.length();
        float theta = acos(delta.y*transducers[t].normal / r);
        float phi = transducers[t].focusedPhase;

        float T2=2 * j1f(SimPoints::WAVE_K * SimPoints::TransducerDiameter/2*sinf(theta)) /
                (SimPoints::WAVE_K*SimPoints::TransducerDiameter/2*sinf(theta));
        float re = SimPoints::P0 * T2 / r * cos(phi+SimPoints::WAVE_K*r);
        float im = SimPoints::P0 * T2 / r * sin(phi+SimPoints::WAVE_K*r);

        reAcc += re;
        imAcc += im;
    }
    //reAcc = xIdx ==20 ? 100 : 0;

    float mag = sqrtf(reAcc*reAcc + imAcc*imAcc);
    values[idx]=mag;


}

extern "C"
void cudaCalc(float *values, Transducer *transducers) {

    auto t1 = now();

    hipExtent extent= make_hipExtent(SimPoints::NumSimPointsX * sizeof(float), SimPoints::NumSimPointsZ, SimPoints::NumSimPointsY);
    hipPitchedPtr d_values;
    hipMalloc3D(&d_values, extent);

    hipMemcpyToSymbol(HIP_SYMBOL(d_transducers), transducers, SimPoints::NumTransducers * sizeof(Transducer));

    auto t2 = now();

    dim3 blockSize(SimPoints::NumSimPointsY, 1, 1);
    dim3 numBlocks(SimPoints::NumSimPointsX, SimPoints::NumSimPointsZ, 1);

    simKernel<<<numBlocks, blockSize>>>(d_values);
    hipDeviceSynchronize();


    auto t3 = now();

    hipMemcpy3DParms memcpyParms = {0};
    memcpyParms.srcPtr=d_values;
    memcpyParms.dstPtr.ptr=values;
    memcpyParms.dstPtr.pitch=SimPoints::NumSimPointsX *sizeof(float);
    memcpyParms.dstPtr.xsize=SimPoints::NumSimPointsX;
    memcpyParms.dstPtr.ysize=SimPoints::NumSimPointsZ;
    memcpyParms.extent.width=SimPoints::NumSimPointsX * sizeof(float);
    memcpyParms.extent.height=SimPoints::NumSimPointsZ;
    memcpyParms.extent.depth=SimPoints::NumSimPointsY;
    memcpyParms.kind=hipMemcpyDeviceToHost;

    hipError_t ret=hipMemcpy3D(&memcpyParms);


    if(ret!= hipSuccess)
        std::cout << "error " << hipGetErrorString(ret) <<"\n";

    hipFree(d_values.ptr);

    auto t4 = now();

    /*std::cout << "GPU time breakdown--------\n";
    std::cout << "loading into device memory: " << milliseconds(t2 - t1) << " milliseconds\n";
    std::cout << "actual addition:            " << milliseconds(t3 - t2) << " milliseconds\n";
    std::cout << "loading into host memory:   " << milliseconds(t4 - t3) << " milliseconds\n";
*/
    return;
}
